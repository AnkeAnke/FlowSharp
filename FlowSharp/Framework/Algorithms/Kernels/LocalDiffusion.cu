#include "hip/hip_runtime.h"
//Includes for IntelliSense 
#define _SIZE_T_DEFINED
#ifndef __HIPCC__
#define __HIPCC__
#endif
#ifndef __cplusplus
#define __cplusplus
#endif
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <>
#include <>
#include "float.h"
#include <builtin_types.h>
#include <vector_functions.h>
#include <hipblas.h>
#include <hipsparse.h>

texture<float, 2, hipReadModeElementType> vX_t0;
texture<float, 2, hipReadModeElementType> vY_t0;
texture<float, 2, hipReadModeElementType> vX_t1;
texture<float, 2, hipReadModeElementType> vY_t1;
//texture<float, 2, hipReadModeElementType> referenceMap;

extern "C"  {
	__constant__ float Variance = 1.0f;
	// Change those two depending on cut or reference execution.
	__constant__ int Width = 200;
	__constant__ int Height = 200;
	__constant__ int WidthCells = 200;
	__constant__ int HeightCells = 200;
	__constant__ int NumParticles = 1024;
	__constant__ float TimeInGrid = 15.0f / 2.59f;
	__constant__ float IntegrationLength = 1.0f;
	__constant__ float StepSize = 0.3f;
	__constant__ float Invalid = 3600000000;
	__constant__ int CellToSeedRatio = 10;
	__constant__ unsigned int HalfNumNeighbors = 4;
	__device__ const float TWO_PI = 2.0f*3.14159265358979323846f;

	// ~~~~~~~~~~~~~~~~~~~~ Random Functions ~~~~~~~~~~~~~~~~~~~~ //
	__device__ unsigned int WangHash(unsigned int seed)
	{
		seed = (seed ^ 61) ^ (seed >> 16);
		seed *= 9;
		seed = seed ^ (seed >> 4);
		seed *= 0x27d4eb2d;
		seed = seed ^ (seed >> 15);
		return seed;
	}

	__device__ unsigned int Xorshift(unsigned int seed)
	{
		seed = seed ^ (seed << 13); 
		seed = seed ^ (seed >> 17); 
		seed = seed ^ (seed << 5);

		return seed;
	}

	__device__ float RandomWang(unsigned int& seed)
	{
		return (float)(WangHash(seed) % 8388593 + 1) / 8388594.0;
	}

	__device__ float2 BoxMuller(unsigned int& seed)
	{
		float u1 = RandomWang(seed);
		seed = WangHash(seed);
		float u2 = RandomWang(seed);

		float lnU = sqrt(-2.0f * log(u1)) * Variance;
		float piU = TWO_PI * log(u2);
		return make_float2(lnU * cos(piU), lnU * sin(piU));
	}

	// ~~~~~~~~~~~~~~ Particle Advection ~~~~~~~~~~~~~~ //
	__device__ float2 AdvectParticle(float2 seed)
	{
		// Take threadIdx as particleIdx;
		int particleIdx = threadIdx.x;

		float3 pos = make_float3(seed.x, seed.y, 0);
		int numSteps = 1000;

		float3 v = make_float3(0, 0, 0);
		float valid;

		unsigned int rndSeed = seed.x + seed.y*WidthCells + particleIdx;

		// Should I even start integrating? Should never happen, though...
		valid = tex2D(vX_t0, pos.x + 0.5, pos.y + 0.5);
		// Works.
		if (valid < Invalid)
		{
			// Step.
			while (pos.z < IntegrationLength && numSteps-- > 0 && pos.x >= 0 && pos.y >= 0 && (int)(pos.x + 0.5) < Width && (int)(pos.y + 0.5) < Height)
			{
				float t = pos.z / TimeInGrid;

				// t0
				v.x = tex2D(vX_t0, pos.x, pos.y) * (1 - t);
				v.y = tex2D(vY_t0, pos.x, pos.y) * (1 - t);
				// t1
				v.x += tex2D(vX_t1, pos.x, pos.y) * t;
				v.y += tex2D(vY_t1, pos.x, pos.y) * t;
				v.z = 1;

				// Add diffusion.
				float2 gauss = BoxMuller(rndSeed);
				rndSeed = Xorshift(rndSeed);
				v.x += gauss.x;
				v.y += gauss.y;

				// Critical point?
				float vLen = v.x*v.x + v.y*v.y + 1;
				vLen = sqrt(vLen);

				// Bring to step size.
				float3 cpy;// = pos;
				float stride = min(StepSize/vLen, IntegrationLength - pos.z);
				cpy.x = pos.x + v.x * stride;
				cpy.y = pos.y + v.y * stride;
				cpy.z = pos.z + stride;

				// Test the rounded position again. Valid?
				valid = tex2D(vX_t0, (int)(cpy.x + 0.5), (int)(cpy.y + 0.5));
				if (valid == Invalid || cpy.x < 0 || cpy.y < 0 || (int)(cpy.x + 0.5)>= Width || (int)(pos.y + 0.5) >= Height)
				{
					break;
				}
				pos = cpy;
			}
		}
		return make_float2(pos.x, pos.y);
	}


	// Start integrating from a common seed. 
	// For cut particles: Origin should be (0,0) if all blocks can be computed in parallel.
	// For reference particles: Origin is seed point, scaled by ratio.
	__device__ float2 LoadAdvect(int2 origin)
	{
		// Offset position by origin. Assume all blocks are in the seed range.
		int px = origin.x + blockIdx.x;
		int py = origin.y + blockIdx.y;
		float2 position = make_float2(px, py);

		return AdvectParticle(position);
	}

	//// ~~~~~~~~~~~ FTLE Start Settings ~~~~~~~~~~~ //
	//__device__ float2 LoadAdvectFTLE(int2 origin)
	//{
	//	// Offset position by origin. Assume all blocks are in the seed range.
	//	int px = origin.x + blockIdx.x;
	//	int py = origin.y + blockIdx.y;
	//	float2 position = make_float2(px, py);


	//	return AdvectParticle(position);
	//}

	// ~~~~~~~~~~~~ Start Reference Particle Integration ~~~~~~~~~~~~ //
	__global__ void LoadAdvectReference(float2* positions, int2 seed)
	{
		positions[threadIdx.x] = LoadAdvect(seed);
	}

	// ~~~~~~~~~~~~ Start Cut Particle Integration ~~~~~~~~~~~~ //
	__global__ void LoadAdvectCut(float2* positions, int2 origin)
	{
		int2 pos = make_int2(origin.x + blockIdx.x, origin.y + blockIdx.y);
		positions[threadIdx.x + (pos.x + pos.y * Width) * blockDim.x] = LoadAdvect(origin);
	}

	// ~~~~~~~~~~~~ Advect Cut Particle ~~~~~~~~~~~~ //
	__global__ void AdvectCut(float2* positions, int2 origin)
	{
		int idx = origin.x + blockIdx.x + (origin.y + blockIdx.y) * Width;
		idx *= blockDim.x;
		idx += threadIdx.x;
		positions[idx] = AdvectParticle(positions[idx]);
	}

	// ~~~~~~~~~~~~ Advect Reference Particle ~~~~~~~~~~~~ //
	__global__ void AdvectReference(float2* positions)
	{
		positions[threadIdx.x] = AdvectParticle(positions[threadIdx.x]);
	}

	// ~~~~~~~~~~~~ Advect Reference Particles into Array ~~~~~~~~~~~~ //
	__global__ void AdvectStoreReference(float2* positions, float* referenceMap)
	{
		float2 pos = AdvectParticle(positions[threadIdx.x]);
		float* dest = referenceMap + (int)(pos.x * CellToSeedRatio + 0.5) + (int)(pos.y * CellToSeedRatio + 0.5) * WidthCells;
		atomicAdd(dest, 1.0f / blockDim.x);
	}

	// ~~~~~~~~~~~~ Cutting all other Particles with Reference Map ~~~~~~~~~~~~~~ //
	__global__ void FetchSumStoreCut(hipSurfaceObject_t cuts, float2* positions, float* referenceMap)
	{
		__shared__ float scan[1024];
		int idx = blockIdx.x + blockIdx.y * Width;
		idx *= blockDim.x;
		idx += threadIdx.x;
		// !beware! "size mangling" occuring.
		scan[threadIdx.x] = referenceMap[(int)(positions[idx].x * CellToSeedRatio + 0.5) + (int)(positions[idx].y * CellToSeedRatio + 0.5) * WidthCells]; //referenceMap[blockIdx.x*CellToSeedRatio + blockIdx.y * CellToSeedRatio*WidthCells];//

		__syncthreads();

		// Reduce.
		for (int nextSize = blockDim.x / 2; nextSize > 0; nextSize/=2)
		{
			if (threadIdx.x < nextSize)
				scan[threadIdx.x] += scan[threadIdx.x + nextSize];
			__syncthreads();
		}

		// Write data to texture.
		if (threadIdx.x == 0)
		{
			//surf2Dwrite(0.1f, cuts, positions[idx].x * sizeof(float), positions[idx].y, hipBoundaryModeTrap);
			surf2Dwrite(scan[0] / blockDim.x, cuts, blockIdx.x*sizeof(float), blockIdx.y, hipBoundaryModeTrap);
		}
	}

	// ~~~~~~~~~~~~ Copy the Array Data to Texture ~~~~~~~~~~~~ //
	__global__ void ReferenceToTexture(hipSurfaceObject_t referenceTex, float* data)
	{
		int px = blockIdx.x * blockDim.x + threadIdx.x;
		int py = blockIdx.y * blockDim.y + threadIdx.y;
		int linIdx = py * WidthCells + px;
		if (px < WidthCells && py < HeightCells)
		{
			surf2Dwrite(data[linIdx], referenceTex, px*sizeof(float), py, hipBoundaryModeTrap);
			data[linIdx] = 0;
		}
	}

	__global__ void CutNeighbors(float* neighborMap, float2* positions, unsigned int offset, unsigned int neighborOffset, unsigned int neighbor)
	{
			// Enough memory for this?
			__shared__ float2 reference[1024];

			// Adding an offset in case the reference is "left" or "below" the current point.
			int idx = blockIdx.x + blockIdx.y * Width + offset;
			int idxRef = idx + neighborOffset;
			idx *= blockDim.x;
			idxRef *= blockDim.x;

			reference[threadIdx.x] = positions[idxRef + threadIdx.x];
			__syncthreads();	
			
			// Here comes the hammer methode!
			float2 pos = positions[idx + threadIdx.x];
			float sum = 0;

			// Compare to each particle at the reference position.
			for (int ref = 0; ref < blockDim.x; ++ref)
			{
				float diffX = (reference[ref].x - pos.x);
				float diffY = (reference[ref].y - pos.y);
				// Is the reference particle within the same "cell"?
//				if (diffX*diffX < 0.5f && diffY*diffY < 0.5f)
//					sum++;
				sum += 1.0 / max(0.5, sqrt(diffX*diffX + diffY*diffY));
			}

			__syncthreads();

			// Use the same buffer we had before. Reference particles are not needed anymore.
			reference[threadIdx.x].y = (float)sum/blockDim.x;
			__syncthreads();

			// Reduce.
			for (int nextSize = blockDim.x / 2; nextSize > 0; nextSize /= 2)
			{
				if (threadIdx.x < nextSize)
					reference[threadIdx.x].y += reference[threadIdx.x + nextSize].y;
				__syncthreads();
			}

			// Write data to texture.
			if (threadIdx.x == 0)
			{
				//surf2Dwrite(0.1f, cuts, positions[idx].x * sizeof(float), positions[idx].y, hipBoundaryModeTrap);
				//surf2Dwrite(reference[0].x / blockDim.x, cuts, blockIdx.x*sizeof(float), blockIdx.y, hipBoundaryModeTrap);
				neighborMap[(blockIdx.x + blockIdx.y * Width + offset) * HalfNumNeighbors + neighbor] = reference[0].y / blockDim.x;
			}
	}

	__global__ void DeformationTensorFTLE (float* neighborMap, float2* positions, unsigned int offset, unsigned int neighborOffset, unsigned int neighbor)
	{
		if (threadIdx.x > 0)
			return;
		if (blockIdx.x == 0 || blockIdx.x == Width - 1 || blockIdx.y == 0 || blockIdx.y == Height - 1)
			return;

		int idx = blockIdx.x + blockIdx.y * Width;
		int idxP; int idxN;

//		reference[threadIdx.x] = positions[idxRef + threadIdx.x];
		switch (neighbor)
		{
			// Right - Left.
		case 0:
		case 2:
			idxP = idx + 1;
			idxN = idx - 1;
			break;
			// Up - Down.
		case 1:
		case 3:
			idxP = idx + Width;
			idxN = idx - Width;
			break;
		}
		// Write data to texture.
		float diff = neighbor < 2 ?
			// U derivative.
			positions[idxP].x - positions[idxN].x :
			// V derivative.
			positions[idxP].y - positions[idxN].y;
		//surf2Dwrite(0.1f, cuts, positions[idx].x * sizeof(float), positions[idx].y, hipBoundaryModeTrap);
		//surf2Dwrite(reference[0].x / blockDim.x, cuts, blockIdx.x*sizeof(float), blockIdx.y, hipBoundaryModeTrap);
		neighborMap[idx * HalfNumNeighbors + neighbor] = diff;	
	}

	__global__ void ScanStoreDensity(hipSurfaceObject_t dens, float* neighborMap, unsigned int pad, int2 origin)
	{
			int px = blockIdx.x * blockDim.x + threadIdx.x + origin.x;
			int py = blockIdx.y * blockDim.y + threadIdx.y + origin.y;
			int linIdx = py * Width + px;
			linIdx *= HalfNumNeighbors;
			// Exclude outermost pixels.
			if (px > 0 && py > 0 && px < Width-1 && py < Height-1)
			{
				float density = 0;
				// Right.
				density += neighborMap[linIdx + 0];
				// Left.
				density += neighborMap[linIdx - HalfNumNeighbors + 0];

				// Up.
				density += neighborMap[linIdx + 1];
				// Down.
				density += neighborMap[linIdx - HalfNumNeighbors*Width + 1];

				// Upper Right.
				density += neighborMap[linIdx + 2];
				density += neighborMap[linIdx - HalfNumNeighbors*(Width + 1) + 2];

				// Upper Left.
				density += neighborMap[linIdx + 3];
				density += neighborMap[linIdx + HalfNumNeighbors*(1-Width) + 3];

				surf2Dwrite(density/8, dens, px * sizeof(float), py, hipBoundaryModeTrap);
			}
	}

	__global__ void ScanStoreMin(hipSurfaceObject_t mins, float* neighborMap, unsigned int pad, int2 origin)
	{
		int px = blockIdx.x * blockDim.x + threadIdx.x + origin.x;
		int py = blockIdx.y * blockDim.y + threadIdx.y + origin.y;
		int linIdx = py * Width + px;
		linIdx *= HalfNumNeighbors;
		// Exclude outermost pixels.
		if (px > 0 && py > 0 && px < Width - 1 && py < Height - 1)
		{
			float density = 1;
			// Right.
			density = min(neighborMap[linIdx + 0], density);
			// Left.
			density = min(neighborMap[linIdx - HalfNumNeighbors + 0], density);

			// Up.
			density = min(neighborMap[linIdx + 1], density);
			// Down.
			density = min(neighborMap[linIdx - HalfNumNeighbors*Width + 1], density);

			// Upper Right.
			density = min(neighborMap[linIdx + 2], density);
			density = min(neighborMap[linIdx - HalfNumNeighbors*(Width + 1) + 2], density);

			// Upper Left.
			density = min(neighborMap[linIdx + 3], density);
			density = min(neighborMap[linIdx + HalfNumNeighbors*(1 - Width) + 3], density);

			surf2Dwrite(density, mins, px * sizeof(float), py, hipBoundaryModeTrap);
		}
	}

	__global__ void ScanStoreMax(hipSurfaceObject_t maxs, float* neighborMap, unsigned int pad, int2 origin)
	{
		int px = blockIdx.x * blockDim.x + threadIdx.x + origin.x;
		int py = blockIdx.y * blockDim.y + threadIdx.y + origin.x;
		int linIdx = py * Width + px;
		linIdx *= HalfNumNeighbors;
		// Exclude outermost pixels.
		if (px > 0 && py > 0 && px < Width - 1 && py < Height - 1)
		{
			float density = 0;
			// Right.
			density = max(neighborMap[linIdx + 0], density);
			// Left.
			density = max(neighborMap[linIdx - HalfNumNeighbors + 0], density);

			// Up.
			density = max(neighborMap[linIdx + 1], density);
			// Down.
			density = max(neighborMap[linIdx - HalfNumNeighbors*Width + 1], density);

			// Upper Right.
			density = max(neighborMap[linIdx + 2], density);
			density = max(neighborMap[linIdx - HalfNumNeighbors*(Width + 1) + 2], density);

			// Upper Left.
			density = max(neighborMap[linIdx + 3], density);
			density = max(neighborMap[linIdx + HalfNumNeighbors*(1 - Width) + 3], density);

			surf2Dwrite(density, maxs, px * sizeof(float), py, hipBoundaryModeTrap);
		}
	}

	__global__ void ScanStoreRange(hipSurfaceObject_t diffs, float* neighborMap, unsigned int pad, int2 origin)
	{
		int px = blockIdx.x * blockDim.x + threadIdx.x + origin.x;
		int py = blockIdx.y * blockDim.y + threadIdx.y + origin.x;
		int linIdx = py * Width + px;
		linIdx *= HalfNumNeighbors;

		if (px > 0 && py > 0 && px < Width - 1 && py < Height - 1)
		{
			float minDens = 1;
			// Right.
			minDens = min(neighborMap[linIdx + 0], minDens);
			// Left.
			minDens = min(neighborMap[linIdx - HalfNumNeighbors + 0], minDens);

			// Up.
			minDens = min(neighborMap[linIdx + 1], minDens);
			// Down.
			minDens = min(neighborMap[linIdx - HalfNumNeighbors*Width + 1], minDens);

			// Upper Right.
			minDens = min(neighborMap[linIdx + 2], minDens);
			minDens = min(neighborMap[linIdx - HalfNumNeighbors*(Width + 1) + 2], minDens);

			// Upper Left.
			minDens = min(neighborMap[linIdx + 3], minDens);
			minDens = min(neighborMap[linIdx + HalfNumNeighbors*(1 - Width) + 3], minDens);

			float maxDens = 0;
			// Right.
			maxDens = max(neighborMap[linIdx + 0], maxDens);
			// Left.
			maxDens = max(neighborMap[linIdx - HalfNumNeighbors + 0], maxDens);

			// Up.
			maxDens = max(neighborMap[linIdx + 1], maxDens);
			// Down.
			maxDens = max(neighborMap[linIdx - HalfNumNeighbors*Width + 1], maxDens);

			// Upper Right.
			maxDens = max(neighborMap[linIdx + 2], maxDens);
			maxDens = max(neighborMap[linIdx - HalfNumNeighbors*(Width + 1) + 2], maxDens);

			// Upper Left.
			maxDens = max(neighborMap[linIdx + 3], maxDens);
			maxDens = max(neighborMap[linIdx + HalfNumNeighbors*(1 - Width) + 3], maxDens);

			surf2Dwrite(maxDens - minDens, diffs, px * sizeof(float), py, hipBoundaryModeTrap);
		}
	}

	__global__ void ScanStoreDirection(hipSurfaceObject_t map, float* neighborMap, unsigned int neighbor, int2 origin)
	{
		int px = blockIdx.x * blockDim.x + threadIdx.x + origin.x;
		int py = blockIdx.y * blockDim.y + threadIdx.y + origin.x;
		int linIdx = py * Width + px;
		linIdx *= HalfNumNeighbors;
		// Exclude outermost pixels.
		if (px > 0 && py > 0 && px < Width - 1 && py < Height - 1)
		{
			float density = 0;
			switch (neighbor)
			{
			case 0:
			case 4:
				// Right.
				density += neighborMap[linIdx + 0];
				// Left.
				density += neighborMap[linIdx - HalfNumNeighbors + 0];
				break;
			case 1:
			case 5:
				// Up.
				density += neighborMap[linIdx + 1];
				// Down.
				density += neighborMap[linIdx - HalfNumNeighbors*Width + 1];
				break;
			case 2:
			case 6:
				// Upper Right.
				density += neighborMap[linIdx + 2];
				density += neighborMap[linIdx - HalfNumNeighbors*(Width + 1) + 2];
				break;
			case 3:
			default:
				// Upper Left.
				density += neighborMap[linIdx + 3];
				density += neighborMap[linIdx + HalfNumNeighbors*(1 - Width) + 3];
				break;
			}
			surf2Dwrite(density, map, px * sizeof(float), py, hipBoundaryModeTrap);
		}
	}

	__global__ void ScanStoreNeighbor(hipSurfaceObject_t map, float* neighborMap, unsigned int neighbor, int2 origin)
	{
		int px = blockIdx.x * blockDim.x + threadIdx.x + origin.x;
		int py = blockIdx.y * blockDim.y + threadIdx.y + origin.x;
		int linIdx = py * Width + px;
		linIdx *= HalfNumNeighbors;
		// Exclude outermost pixels.
		if (px > 0 && py > 0 && px < Width - 1 && py < Height - 1)
		{
			float density = 0;
			switch (neighbor)
			{
			case 0:
				// Right.
				density = neighborMap[linIdx + 0];
				break;
			case 4:
				// Left.
				density = neighborMap[linIdx - HalfNumNeighbors + 0];
				break;
			case 1:
				// Up.
				density = neighborMap[linIdx + 1];
				break;
			case 5:
				// Down.
				density = neighborMap[linIdx - HalfNumNeighbors*Width + 1];
				break;
			case 2:
				// Upper Right.
				density = neighborMap[linIdx + 2];
				break;
			case 6:
				// Lower left.
				density = neighborMap[linIdx - HalfNumNeighbors*(Width + 1) + 2];
				break;
			case 3:
				// Upper Left.
				density = neighborMap[linIdx + 3];
				break;
			default:
				// Lower Right.
				density = neighborMap[linIdx + HalfNumNeighbors*(1 - Width) + 3];
				break;
			}
			surf2Dwrite(density, map, px * sizeof(float), py, hipBoundaryModeTrap);
		}
	}

	__global__ void ScanStoreFTLE(hipSurfaceObject_t map, float* neighborMap, unsigned int neighbor, int2 origin)
	{
		int px = blockIdx.x * blockDim.x + threadIdx.x;
		int py = blockIdx.y * blockDim.y + threadIdx.y;
		int linIdx = py * Width + px;
		linIdx *= HalfNumNeighbors;
		// Exclude outermost pixels.
		if (px > 0 && py > 0 && px < Width - 1 && py < Height - 1)
		{
			// Compute Eigenvalues.
			// Load 4 values.
			float Ux = neighborMap[linIdx + 0];
			float Uy = neighborMap[linIdx + 1];
			float Vx = neighborMap[linIdx + 2];
			float Vy = neighborMap[linIdx + 3];
			float a = Ux*Ux + Vx*Vx;
			float b = Ux*Uy + Vx*Vy;
			float d = Uy*Uy + Vy*Vy;

			// Helpers.
			float Th = (a - d) * 0.5f;
			float D = a * d - b * b;
			float root = Th * Th - D;

			root = max(0.0f, root);

			root = sqrt(root);
			float l0 = Th + root;
			float l1 = Th - root;

			float lambdaMax = max(0.000001, max(l0, l1));

			//float a = neighborMap[linIdx + 0];
			//float b = neighborMap[linIdx + 1];
			//float c = neighborMap[linIdx + 2];
			//float d = neighborMap[linIdx + 3];

			//a *= a;
			//float bc = b*b * c*c;
			//d *= d;

			//float root = a*a - 2 * a*d + 4 * bc + d*d;
			//root = sqrt(max(0.0, root));
			//float lambdaMax = (a + d) * 0.5;
			surf2Dwrite(logf(sqrt(lambdaMax)) / IntegrationLength, map, px * sizeof(float), py, hipBoundaryModeTrap);
		}
	}
}