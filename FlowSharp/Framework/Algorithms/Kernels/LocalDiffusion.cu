#include "hip/hip_runtime.h"
//Includes for IntelliSense 
#define _SIZE_T_DEFINED
#ifndef __HIPCC__
#define __HIPCC__
#endif
#ifndef __cplusplus
#define __cplusplus
#endif
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <>
#include <>
#include "float.h"
#include <builtin_types.h>
#include <vector_functions.h>
#include <hipblas.h>
#include <hipsparse.h>

texture<float, 2, hipReadModeElementType> vX_t0;
texture<float, 2, hipReadModeElementType> vY_t0;
texture<float, 2, hipReadModeElementType> vX_t1;
texture<float, 2, hipReadModeElementType> vY_t1;
//texture<float, 2, hipReadModeElementType> referenceMap;

extern "C"  {
	__constant__ float Variance = 1.0f;
	// Change those two depending on cut or reference execution.
	__constant__ int Width = 200;
	__constant__ int Height = 200;
	__constant__ int WidthCells = 200;
	__constant__ int HeightCells = 200;
	__constant__ int NumParticles = 1024;
	__constant__ float TimeInGrid = 15.0f / 2.59f;
	__constant__ float IntegrationLength = 1.0f;
	__constant__ float StepSize = 0.3f;
	__constant__ float Invalid = 3600000000;
	__constant__ int CellToSeedRatio = 10;
	__device__ const float TWO_PI = 2.0f*3.14159265358979323846f;

	// ~~~~~~~~~~~~~~~~~~~~ Random Functions ~~~~~~~~~~~~~~~~~~~~ //
	__device__ unsigned int WangHash(unsigned int seed)
	{
		seed = (seed ^ 61) ^ (seed >> 16);
		seed *= 9;
		seed = seed ^ (seed >> 4);
		seed *= 0x27d4eb2d;
		seed = seed ^ (seed >> 15);
		return seed;
	}

	__device__ unsigned int Xorshift(unsigned int seed)
	{
		seed = seed ^ (seed << 13); 
		seed = seed ^ (seed >> 17); 
		seed = seed ^ (seed << 5);

		return seed;
	}

	__device__ float RandomWang(unsigned int& seed)
	{
		return (float)(WangHash(seed) % 8388593 + 1) / 8388594.0;
	}

	__device__ float2 BoxMuller(unsigned int& seed)
	{
		float u1 = RandomWang(seed);
		seed = WangHash(seed);
		float u2 = RandomWang(seed);

		float lnU = sqrt(-2.0f * log(u1)) * Variance;
		float piU = TWO_PI * log(u2);
		return make_float2(lnU * cos(piU), lnU * sin(piU));
	}

	// ~~~~~~~~~~~~~~ Particle Advection ~~~~~~~~~~~~~~ //
	__device__ float2 AdvectParticle(float2 seed)
	{
		// Take threadIdx as particleIdx;
		int particleIdx = threadIdx.x;

		float3 pos = make_float3(seed.x, seed.y, 0);
		int numSteps = 100000;

		float3 v = make_float3(0, 0, 0);
		float valid;

		unsigned int rndSeed = seed.x + seed.y*WidthCells + particleIdx;

		// Should I even start integrating? Should never happen, though...
		valid = tex2D(vX_t0, pos.x + 0.5, pos.y + 0.5);
		// Works.
		if (valid < Invalid)
		{
			// Step.
			while (pos.z < IntegrationLength && numSteps-- > 0 && pos.x >= 0 && pos.y >= 0 && (int)(pos.x + 0.5) < Width && (int)(pos.y + 0.5) < Height)
			{
				float t = pos.z / TimeInGrid;

				// t0
				v.x = tex2D(vX_t0, pos.x, pos.y) * (1 - t);
				v.y = tex2D(vY_t0, pos.x, pos.y) * (1 - t);
				// t1
				v.x += tex2D(vX_t1, pos.x, pos.y) * t;
				v.y += tex2D(vY_t1, pos.x, pos.y) * t;
				v.z = 1;

				// Add diffusion.
				float2 gauss = BoxMuller(rndSeed);
				rndSeed = Xorshift(rndSeed);
				v.x += gauss.x;
				v.y += gauss.y;

				//// Critical point?
				float vLen = v.x*v.x + v.y*v.y + 1;
				vLen = sqrt(vLen);

				// Bring to step size.
				float3 cpy;// = pos;
				float stride = min(StepSize/vLen, IntegrationLength - pos.z);
				cpy.x = pos.x + v.x * stride;
				cpy.y = pos.y + v.y * stride;
				cpy.z = pos.z + stride;

				// Test the rounded position again. Valid?
				valid = tex2D(vX_t0, (int)(cpy.x + 0.5), (int)(cpy.y + 0.5));
				if (valid == Invalid || cpy.x < 0 || cpy.y < 0 || (int)(cpy.x + 0.5)>= Width || (int)(pos.y + 0.5) >= Height)
				{
					break;
				}
				pos = cpy;
			}
		}
		return make_float2(pos.x, pos.y);
	}


	// Start integrating from a common seed. 
	// For cut particles: Origin should be (0,0) if all blocks can be computed in parallel.
	// For reference particles: Origin is seed point, scaled by ratio.
	__device__ float2 LoadAdvect(int2 origin)
	{
		// Offset position by origin. Assume all blocks are in the seed range.
		int px = origin.x + blockIdx.x;
		int py = origin.y + blockIdx.y;
		float2 position = make_float2(px, py);

		return AdvectParticle(position);
	}

	// ~~~~~~~~~~~~ Start Reference Particle Integration ~~~~~~~~~~~~ //
	__global__ void LoadAdvectReference(float2* positions, int2 seed)
	{
		positions[threadIdx.x] = LoadAdvect(seed);
	}

	// ~~~~~~~~~~~~ Start Cut Particle Integration ~~~~~~~~~~~~ //
	__global__ void LoadAdvectCut(float2* positions, int2 origin)
	{
		int2 pos = make_int2(origin.x + blockIdx.x, origin.y + blockIdx.y);
		positions[threadIdx.x + (pos.x + pos.y * Width) * blockDim.x] = LoadAdvect(origin);
	}

	// ~~~~~~~~~~~~ Advect Cut Particle ~~~~~~~~~~~~ //
	__global__ void AdvectCut(float2* positions, int2 origin)
	{
		int idx = origin.x + blockIdx.x + (origin.y + blockIdx.y) * Width;
		idx *= blockDim.x;
		idx += threadIdx.x;
		positions[idx] = AdvectParticle(positions[idx]);
	}

	// ~~~~~~~~~~~~ Advect Reference Particle ~~~~~~~~~~~~ //
	__global__ void AdvectReference(float2* positions)
	{
		positions[threadIdx.x] = AdvectParticle(positions[threadIdx.x]);
	}

	// ~~~~~~~~~~~~ Advect Reference Particles into Array ~~~~~~~~~~~~ //
	__global__ void AdvectStoreReference(float2* positions, float* referenceMap)
	{
		float2 pos = AdvectParticle(positions[threadIdx.x]);
		float* dest = referenceMap + (int)(pos.x * CellToSeedRatio + 0.5) + (int)(pos.y * CellToSeedRatio + 0.5) * WidthCells;
		atomicAdd(dest, 1.0f / blockDim.x);
	}

	// ~~~~~~~~~~~~ Cutting all other Particles with Reference Map ~~~~~~~~~~~~~~ //
	__global__ void FetchSumStoreCut(hipSurfaceObject_t cuts, float2* positions, float* referenceMap)
	{
		__shared__ float scan[1024];
		int idx = blockIdx.x + blockIdx.y * Width;
		idx *= blockDim.x;
		idx += threadIdx.x;
		// !beware! "size mangling" occuring.
		scan[threadIdx.x] = referenceMap[(int)(positions[idx].x * CellToSeedRatio + 0.5) + (int)(positions[idx].y * CellToSeedRatio + 0.5) * WidthCells]; //referenceMap[blockIdx.x*CellToSeedRatio + blockIdx.y * CellToSeedRatio*WidthCells];//

		__syncthreads();

		// Reduce.
		for (int nextSize = blockDim.x / 2; nextSize > 0; nextSize/=2)
		{
			if (threadIdx.x < nextSize)
				scan[threadIdx.x] += scan[threadIdx.x + nextSize];
			__syncthreads();
		}

		// Write data to texture.
		if (threadIdx.x == 0)
		{
			//surf2Dwrite(0.1f, cuts, positions[idx].x * sizeof(float), positions[idx].y, hipBoundaryModeTrap);
			surf2Dwrite(scan[0] / blockDim.x, cuts, blockIdx.x*sizeof(float), blockIdx.y, hipBoundaryModeTrap);
		}
	}

	// ~~~~~~~~~~~~ Copy the Array Data to Texture ~~~~~~~~~~~~ //
	__global__ void ReferenceToTexture(hipSurfaceObject_t referenceTex, float* data)
	{
		int px = blockIdx.x * blockDim.x + threadIdx.x;
		int py = blockIdx.y * blockDim.y + threadIdx.y;
		int linIdx = py * WidthCells + px;
		if (px < WidthCells && py < HeightCells)
		{
			surf2Dwrite(data[linIdx], referenceTex, px*sizeof(float), py, hipBoundaryModeTrap);
			data[linIdx] = 0;
		}
	}

	// Gradient Kernels
	// ~~~~~~~~ Cut with right Particle Cloud ~~~~~~~~ //
	__global__ void CutX(float* cuts, float2* positions)
	{
		// Enough memory for this?
		float2 reference[1024];
		int idx = blockIdx.x + blockIdx.y * Width;
		idx *= blockDim.x;
		int idxR = idx + blockDim.x;
		idx += threadIdx.x;

		reference[threadIdx.x] = positions[threadIdx.x];
		__syncthreads();
		
		
		// Here comes the hammer methode!
		float2 pos = positions[idx];
		unsigned int sum = 0;
		for (int ref = 0; ref < blockIdx.x; ++ref)
		{
			// Is the reference particle within the same "cell"?
			if (abs((pos.x - reference[idxR].x) * (pos.y - reference[idxR].y)) < 0.25f)
				sum++;
			//positions[idx].x = referenceMap[(int)(positions[idx].x * CellToSeedRatio + 0.5) + (int)(positions[idx].y * CellToSeedRatio + 0.5) * WidthCells]; //referenceMap[blockIdx.x*CellToSeedRatio + blockIdx.y * CellToSeedRatio*WidthCells];//
		}
		__syncthreads();
		// Use the same buffer we had before. Reference particles are not needed anymore.

		reference[threadIdx.x].x = (float)sum;
		__syncthreads();
		// Reduce.
		for (int nextSize = blockDim.x / 2; nextSize > 0; nextSize /= 2)
		{
			if (threadIdx.x < nextSize)
				reference[idx].x += reference[idx + nextSize].x;
			__syncthreads();
		}

		// Write data to texture.
		if (threadIdx.x == 0)
		{
			//surf2Dwrite(0.1f, cuts, positions[idx].x * sizeof(float), positions[idx].y, hipBoundaryModeTrap);
			//surf2Dwrite(reference[0].x / blockDim.x, cuts, blockIdx.x*sizeof(float), blockIdx.y, hipBoundaryModeTrap);
			cuts[blockIdx.x + blockIdx.y * (Width - 1)] = reference[0].x;
		}
	}
	// ~~~~~~~~ Cut with upper Particle Cloud ~~~~~~~~ //
	__global__ void CutY(float* cuts, float2* positions)
	{

	}

	__global__ void StoreXY(hipSurfaceObject_t gradsX, hipSurfaceObject_t gradsY, float* cutsX, float* cutsY)
	{
		int px = blockIdx.x * blockDim.x + threadIdx.x;
		int py = blockIdx.y * blockDim.y + threadIdx.y;
		int linIdx = py * Width + px;
		if (px < Width-1 && py < Height-1)
		{
			surf2Dwrite(cutsX[linIdx], gradsX, px*sizeof(float), py, hipBoundaryModeTrap);
			surf2Dwrite(cutsY[linIdx], gradsY, px*sizeof(float), py, hipBoundaryModeTrap);
		}
	}
}