#include "hip/hip_runtime.h"
//Includes for IntelliSense 
#define _SIZE_T_DEFINED
#ifndef __HIPCC__
#define __HIPCC__
#endif
#ifndef __cplusplus
#define __cplusplus
#endif
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <>
#include <>
#include "float.h"
#include <builtin_types.h>
#include <vector_functions.h>
#include <hipblas.h>
#include <hipsparse.h>

texture<float, 2, hipReadModeElementType> vX_t0;
texture<float, 2, hipReadModeElementType> vY_t0;
texture<float, 2, hipReadModeElementType> vX_t1;
texture<float, 2, hipReadModeElementType> vY_t1;
//texture<float, 2, hipReadModeElementType> referenceMap;

extern "C"  {
	__constant__ float Variance = 1.0f;
	// Change those two depending on cut or reference execution.
	__constant__ int Width = 200;
	__constant__ int Height = 200;
	__constant__ int WidthCells = 200;
	__constant__ int HeightCells = 200;
	__constant__ int NumParticles = 1024;
	__constant__ float TimeInGrid = 15.0f / 2.59f;
	__constant__ float IntegrationLength = 1.0f;
	__constant__ float StepSize = 0.3f;
	__constant__ float Invalid = 3600000000;
	__constant__ int CellToSeedRatio = 10;
	__device__ const float TWO_PI = 2.0f*3.14159265358979323846f;

	// ~~~~~~~~~~~~~~~~~~~~ Random Functions ~~~~~~~~~~~~~~~~~~~~ //
	__device__ unsigned int WangHash(unsigned int seed)
	{
		seed = (seed ^ 61) ^ (seed >> 16);
		seed *= 9;
		seed = seed ^ (seed >> 4);
		seed *= 0x27d4eb2d;
		seed = seed ^ (seed >> 15);
		return seed;
	}

	__device__ float RandomWang(unsigned int& seed)
	{
		return (float)(WangHash(seed) % 8388593) / 8388593.0;
	}

	__device__ float2 BoxMuller(unsigned int& seed)
	{
		float u1 = RandomWang(seed);
		seed = WangHash(seed);
		float u2 = RandomWang(seed);

		float lnU = sqrt(-2.0f * log(u1)) * Variance;
		float piU = TWO_PI * log(u2);
		return make_float2(lnU * cos(piU), lnU * sin(piU));
	}

	// ~~~~~~~~~~~~~~ Particle Advection ~~~~~~~~~~~~~~ //
	__device__ float2 AdvectParticle(float2 seed)
	{
		// Take threadIdx as particleIdx;
		int particleIdx = threadIdx.x;

		float3 pos = make_float3(seed.x, seed.y, 0);
		int numSteps = 100000;

		float3 v = make_float3(0, 0, 0);
		float valid;

		unsigned int rndSeed = 61 + seed.x + seed.y*WidthCells + particleIdx;

		// Should I even start integrating? Should never happen, though...
		valid = tex2D(vX_t0, pos.x + 0.5, pos.y + 0.5);
		// Works.
		if (valid < Invalid)
		{
			// Step.
			while (pos.z < IntegrationLength && numSteps-- > 0 && pos.x >= 0 && pos.y >= 0 && (int)(pos.x + 0.5) < Width && (int)(pos.y + 0.5) < Height)
			{
				float t = pos.z / TimeInGrid;

				// t0
				v.x = tex2D(vX_t0, pos.x, pos.y) * (1 - t);
				v.y = tex2D(vY_t0, pos.x, pos.y) * (1 - t);
				// t1
				v.x += tex2D(vX_t1, pos.x, pos.y) * t;
				v.y += tex2D(vY_t1, pos.x, pos.y) * t;
				v.z = 1;

				// Add diffusion.
				float2 gauss = BoxMuller(rndSeed);
				rndSeed = WangHash(rndSeed);
				v.x += gauss.x;
				v.y += gauss.y;

				//// Critical point?
				float vLen = v.x*v.x + v.y*v.y + 1;
				vLen = sqrt(vLen);

				// Bring to step size.
				float3 cpy;// = pos;
				cpy.x = pos.x + v.x * StepSize / vLen;
				cpy.y = pos.y + v.y * StepSize / vLen;
				cpy.z = pos.z + StepSize / vLen;

				// Test the rounded position again. Valid?
				valid = tex2D(vX_t0, (int)(cpy.x + 0.5), (int)(cpy.y + 0.5));
				if (valid == Invalid)
				{
					break;
				}
				pos = cpy;
			}
		}
		return make_float2(pos.x, pos.y);
	}


	// Start integrating from a common seed. 
	// For cut particles: Origin should be (0,0) if all blocks can be computed in parallel.
	// For reference particles: Origin is seed point, scaled by ratio.
	__device__ float2 LoadAdvect(int2 origin)
	{
		// Offset position by origin. Assume all blocks are in the seed range.
		int px = origin.x + blockIdx.x;
		int py = origin.y + blockIdx.y;
		float2 position = make_float2(px, py);

		return AdvectParticle(position);
	}

	// ~~~~~~~~~~~~ Start Reference Particle Integration ~~~~~~~~~~~~ //
	__global__ void LoadAdvectReference(float2* positions, int2 seed)
	{
		positions[threadIdx.x] = LoadAdvect(seed);
	}

	// ~~~~~~~~~~~~ Start Cut Particle Integration ~~~~~~~~~~~~ //
	__global__ void LoadAdvectCut(float2* positions, int2 origin)
	{
		int2 pos = make_int2(origin.x + blockIdx.x, origin.y + blockIdx.y);
		positions[threadIdx.x + (pos.x + pos.y * Width) * blockDim.x] = LoadAdvect(origin);
	}

	// ~~~~~~~~~~~~ Advect Cut Particle ~~~~~~~~~~~~ //
	__global__ void AdvectCut(float2* positions, int2 origin)
	{
		int idx = origin.x + blockIdx.x + (origin.y + blockIdx.y) * Width;
		idx *= blockDim.x;
		idx += threadIdx.x;
		positions[idx] = AdvectParticle(positions[idx]);
	}

	// ~~~~~~~~~~~~ Advect Reference Particle ~~~~~~~~~~~~ //
	__global__ void AdvectReference(float2* positions)
	{
		positions[threadIdx.x] = AdvectParticle(positions[threadIdx.x]);
	}

	// ~~~~~~~~~~~~ Advect Reference Particles into Array ~~~~~~~~~~~~ //
	__global__ void AdvectStoreReference(float2* positions, float* referenceMap)
	{
		float2 pos = AdvectParticle(positions[threadIdx.x]);
		float* dest = referenceMap + (int)(pos.x * CellToSeedRatio + 0.5) + (int)(pos.y * CellToSeedRatio + 0.5) * WidthCells;
		atomicAdd(dest, 1.0f / blockDim.x);
	}

	// ~~~~~~~~~~~~ Cutting all other Particles with Reference Map ~~~~~~~~~~~~~~ //
	__global__ void FetchSumStoreCut(hipSurfaceObject_t cuts, float2* positions, float* referenceMap)
	{

		int idx = blockIdx.x + blockIdx.y * Width;
		idx *= blockDim.x;
		idx += threadIdx.x;
		// !beware! "size mangling" occuring.
		positions[idx].x = referenceMap[(int)(positions[idx].x * CellToSeedRatio + 0.5) + (int)(positions[idx].y * CellToSeedRatio + 0.5) * WidthCells]; //referenceMap[blockIdx.x*CellToSeedRatio + blockIdx.y * CellToSeedRatio*WidthCells];//

		__syncthreads();

		// Reduce.
		for (int nextSize = blockDim.x / 2; nextSize > 0; nextSize/=2)
		{
			if (threadIdx.x < nextSize)
				positions[idx].x += positions[idx + nextSize].x;
			__syncthreads();
		}

		// Write data to texture.
		if (threadIdx.x == 0)
		{
			//surf2Dwrite(0.1f, cuts, positions[idx].x * sizeof(float), positions[idx].y, hipBoundaryModeTrap);
			surf2Dwrite(positions[idx].x/blockDim.x, cuts, blockIdx.x*sizeof(float), blockIdx.y, hipBoundaryModeTrap);
		}
	}

	// ~~~~~~~~~~~~ Copy the Array Data to Texture ~~~~~~~~~~~~ //
	__global__ void ReferenceToTexture(hipSurfaceObject_t referenceTex, float* data)
	{
		int px = blockIdx.x * blockDim.x + threadIdx.x;
		int py = blockIdx.y * blockDim.y + threadIdx.y;
		int linIdx = py * WidthCells + px;
		if (px < WidthCells && py < HeightCells)
		{
			surf2Dwrite(data[linIdx], referenceTex, px*sizeof(float), py, hipBoundaryModeTrap);
			data[linIdx] = 0;
		}
	}
}










//
////Includes for IntelliSense 
//#define _SIZE_T_DEFINED
//#ifndef __HIPCC__
//#define __HIPCC__
//#endif
//#ifndef __cplusplus
//#define __cplusplus
//#endif
//#include "hip/hip_runtime.h"
//#include ""
//#include <hip/hip_runtime.h>
//#include <>
//#include <>
//#include "float.h"
//#include <builtin_types.h>
//#include <vector_functions.h>
//#include <hipblas.h>
//#include <hipsparse.h>
//
//texture<float, 2, hipReadModeElementType> vX_t0;
//texture<float, 2, hipReadModeElementType> vY_t0;
//texture<float, 2, hipReadModeElementType> vX_t1;
//texture<float, 2, hipReadModeElementType> vY_t1;
//texture<float, 2, hipReadModeElementType> selectionMap;
//
//extern "C"  {
//	__constant__ float Variance = 1.0f;
//	__constant__ int Width = 200;
//	__constant__ int Height = 200;
//	__constant__ int NumParticles = 1024;
//	__constant__ float TimeInGrid = 15.0f / 2.59f;
//	__constant__ float IntegrationLength = 1.0f;
//	__constant__ float StepSize = 0.3f;
//	__constant__ float Invalid = 3600000000;
//	__constant__ int CellToSeedRatio = 10;
//	__device__ const float TWO_PI = 2.0f*3.14159265358979323846f;
//
//	__device__ unsigned int WangHash(unsigned int seed)
//	{
//		seed = (seed ^ 61) ^ (seed >> 16);
//		seed *= 9;
//		seed = seed ^ (seed >> 4);
//		seed *= 0x27d4eb2d;
//		seed = seed ^ (seed >> 15);
//		return seed;
//	}
//
//	__device__ unsigned int RandomUInt(unsigned int& seed)
//	{
//		// Xorshift32
//		seed ^= (seed << 13);
//		seed ^= (seed >> 17);
//		seed ^= (seed << 5);
//
//		return seed;
//	}
//
//	__device__ float Random(unsigned int& seed)
//	{
//		return float(RandomUInt(seed) % 8388593) / 8388593.0;
//	}
//
//	__device__ float RandomWang(unsigned int& seed)
//	{
//		return (float)(WangHash(seed) % 8388593) / 8388593.0;
//	}
//
//	__device__ float2 Random2(unsigned int& seed)
//	{
//		return make_float2(Random(seed), Random(seed));
//	}
//
//
//	// Take idx's as seedc.
//	__device__ float SimpleRandom(float rnd)
//	{
//		float idxDot = threadIdx.x * 12.9898f + blockIdx.x * 78.233f;
//		float val = sin(idxDot*rnd) * 43758.5453f;
//		return val - truncf(val);
//	}
//
//	__device__ float2 BoxMuller(unsigned int& seed)
//	{
//		float u1 = RandomWang(seed);// gauss.x;
//		seed = WangHash(seed);
//		float u2 = RandomWang(seed);// gauss.y;
//
//		float lnU = sqrt(-2.0f * log(u1)) * Variance;
//		float piU = TWO_PI * log(u2);
//		return make_float2(lnU * cos(piU), lnU * sin(piU));
//	}
//
//	__device__ float3 AdvectParticle(int2 seed)
//	{
//		// Take threadIdx as particleIdx;
//		int particleIdx = threadIdx.x;
//
//		float3 pos = make_float3(seed.x, seed.y, 0);
//		int numSteps = 100000;
//
//		float3 v = make_float3(0, 0, 0);
//		float valid;
//
//		unsigned int rndSeed = seed.x + seed.y + particleIdx;
//
//		// Should I even start integrating? Should never happen, though...
//		valid = tex2D(vX_t0, pos.x + 0.5, pos.y + 0.5);
//		// Works.
//		if (valid < Invalid)
//		{
//			// Step.
//			while (pos.z < IntegrationLength && numSteps-- > 0 && pos.x >= 0 && pos.y >= 0 && (int)(pos.x + 0.5) < Width && (int)(pos.y + 0.5) < Height)
//			{
//				float t = pos.z / TimeInGrid;
//
//				// t0
//				v.x = tex2D(vX_t0, pos.x, pos.y) * (1 - t);
//				v.y = tex2D(vY_t0, pos.x, pos.y) * (1 - t);
//				// t1
//				v.x += tex2D(vX_t1, pos.x, pos.y) * t;
//				v.y += tex2D(vY_t1, pos.x, pos.y) * t;
//				v.z = 1;
//
//				// Add diffusion.
//				float2 gauss = BoxMuller(rndSeed);
//				rndSeed = WangHash(rndSeed);
//				v.x += gauss.x;
//				v.y += gauss.y;
//
//				//// Critical point?
//				float vLen = v.x*v.x + v.y*v.y + 1;
//				vLen = sqrt(vLen);
//				//if (vLen < 0.00000001)
//				//{
//				//	break;
//				//}
//
//
//				// Bring to step size.
//				float3 cpy;// = pos;
//				cpy.x = pos.x + v.x * StepSize / vLen;
//				cpy.y = pos.y + v.y * StepSize / vLen;
//				cpy.z = pos.z + StepSize / vLen;
//
//				// Test the rounded position again. Valid?
//				valid = tex2D(vX_t0, (int)(cpy.x + 0.5), (int)(cpy.y + 0.5));
//				if (valid == Invalid)
//				{
//					break;
//				}
//				pos = cpy;
//			}
//		}
//		return pos;
//	}
//
//	__global__ void AdvectSelectionMap(float2* particles, int2 seed)
//	{
//		float3 newPos = AdvectParticle(seed);
//		//float2* writeTo = particles + (int)(newPos.x * CellToSeedRatio + 0.5) + (int)(newPos.y * CellToSeedRatio + 0.5) * Width * CellToSeedRatio;
//		//atomicAdd(writeTo, 1.0f / blockDim.x);
//		particles[threadIdx.x] = make_float2(newPos.x, newPos.y);
//	}
//
//	__global__ void WriteParticlesAtomic(float2* particles, float* data)
//	{
//		float2 pos = particles[threadIdx.x];
//		float* writeTo = data + (int)(pos.x * CellToSeedRatio + 0.5) + (int)(pos.y * CellToSeedRatio + 0.5) * Width * CellToSeedRatio;
//		atomicAdd(writeTo, 1.0f / blockDim.x);
//	}
//
//	// data == mapT1 from step.
//	// mapT1 == mapT0 bound as surface.
//	__global__ void CopySelectionMap(hipSurfaceObject_t mapT1, float* data)
//	{
//		int px = blockIdx.x * blockDim.x + threadIdx.x;
//		int py = blockIdx.y * blockDim.y + threadIdx.y;
//		int linIdx = py * Width * CellToSeedRatio + px;
//		if (px < Width*CellToSeedRatio && py < Height*CellToSeedRatio)
//		{
//			surf2Dwrite(data[linIdx], mapT1, px*sizeof(float), py, hipBoundaryModeTrap);
//			data[linIdx] = 0;
//		}
//	}
//
//	__global__ void AdvectAndCutSeeds(hipSurfaceObject_t referenceTex, int2 origin)
//	{
//		extern __shared__ float cut[];
//		int px = blockIdx.x + origin.x;
//		int py = blockIdx.y + origin.y;
//
//		int2 pos = make_int2(px, py);
//		float3 newPos = AdvectParticle(pos);
//
//		cut[threadIdx.x] = tex2D(selectionMap, newPos.x*CellToSeedRatio, newPos.y*CellToSeedRatio);
//
//		int step = blockDim.x / 2;
//		__syncthreads();
//		for (int vals = step; vals > 0; vals /= 2)
//		{
//			if (threadIdx.x < vals)
//			{
//				int pair = threadIdx.x + vals;
//				cut[threadIdx.x] += cut[pair];
//			}
//			__syncthreads();
//		}
//		//mapT1[px + py * Width] = cut[0] / blockDim.x;
//		if (px < Width && py < Height)
//			surf2Dwrite(cut[0] / blockDim.x, referenceTex, px*sizeof(float), py, hipBoundaryModeTrap);
//
//	}
//
//	//__global__ void CopyCutMap(hipSurfaceObject_t mapT1, float* data)
//	//{
//	//	int px = blockIdx.x * blockDim.x + threadIdx.x;
//	//	int py = blockIdx.y * blockDim.y + threadIdx.y;
//	//	int linIdx = py * Width + px;
//	//	if (px < Width && py < Height)
//	//	{
//	//		surf2Dwrite(data[linIdx], mapT1, px*sizeof(float), py, hipBoundaryModeTrap);
//	//		data[linIdx] = 0;
//	//	}
//	//}
//}